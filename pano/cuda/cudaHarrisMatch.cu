#include "hip/hip_runtime.h"
#include <common.hpp>
#include <limits>
#include <matcher.hpp>
#include <opencv2/core/types.hpp>
#include <sys/types.h>
#include <vector>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,        \
              hipGetErrorString(error));                                      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

__constant__ int cuPatchSize;
__constant__ uint64_t cuMaxSSDThreash;

texture<uchar4, 2, hipReadModeElementType> texImage1;
texture<uchar4, 2, hipReadModeElementType> texImage2;

void setupTexture(hipArray *cuArray1, hipArray *cuArray2) {
  texImage1.addressMode[0] = hipAddressModeWrap;
  texImage1.addressMode[1] = hipAddressModeWrap;
  texImage1.filterMode = hipFilterModePoint;
  texImage1.normalized = false;

  texImage2.addressMode[0] = hipAddressModeWrap;
  texImage2.addressMode[1] = hipAddressModeWrap;
  texImage2.filterMode = hipFilterModePoint;
  texImage2.normalized = false;

  hipBindTextureToArray(texImage1, cuArray1);
  hipBindTextureToArray(texImage2, cuArray2);
}

CudaHarrisKeypointMatcher::CudaHarrisKeypointMatcher(
    cv::Mat &image1, cv::Mat &image2, HarrisCornerOptions options)
    : image1_(image1), image2_(image2), options_(options) {
  auto cudaChennelDesc = hipCreateChannelDesc<uchar4>();
  hipArray *cuArray1, *cuArray2;
  hipMallocArray(&cuArray1, &cudaChennelDesc, image1_.cols, image1_.rows);
  hipMallocArray(&cuArray2, &cudaChennelDesc, image2_.cols, image2_.rows);

  uchar4 *convertedImage1 = new uchar4[image1_.cols * image1_.rows];
  for (int i = 0; i < image1_.rows; ++i) {
    for (int j = 0; j < image1_.cols; ++j) {
      cv::Vec3b pixel = image1_.at<cv::Vec3b>(i, j);
      convertedImage1[i * image1_.cols + j] =
          make_uchar4(pixel[0], pixel[1], pixel[2], 0);
    }
  }
  hipMemcpyToArray(cuArray1, 0, 0, convertedImage1,
                    image1_.cols * image1_.rows * sizeof(uchar4),
                    hipMemcpyHostToDevice);
  delete convertedImage1;

  uchar4 *convertedImage2 = new uchar4[image2_.cols * image2_.rows];
  for (int i = 0; i < image2_.rows; ++i) {
    for (int j = 0; j < image2_.cols; ++j) {
      cv::Vec3b pixel = image2_.at<cv::Vec3b>(i, j);
      convertedImage2[i * image2_.cols + j] =
          make_uchar4(pixel[0], pixel[1], pixel[2], 0);
    }
  }
  hipMemcpyToArray(cuArray2, 0, 0, convertedImage2,
                    image2_.cols * image2_.rows * sizeof(uchar4),
                    hipMemcpyHostToDevice);
  delete convertedImage2;

  setupTexture(cuArray1, cuArray2);
}

CudaHarrisKeypointMatcher::~CudaHarrisKeypointMatcher() {
  hipUnbindTexture(texImage1);
  hipUnbindTexture(texImage2);
}

__global__ void matchKeypointsKernel(const float *kpsL_x, const float *kpsL_y,
                                     const float *kpsR_x, const float *kpsR_y,
                                     int numImage1Rows, int numImage1Cols,
                                     int numImage2Rows, int numImage2Cols,
                                     int numKpsL, int numKpsR,
                                     int *bestMatchIndices,
                                     uint64_t *bestMatchSSDs) {
  // Assuming each block processes one keypoint from keypointsL
  int keypointIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (keypointIdx >= numKpsL)
    return;

  int patchSize = cuPatchSize;

  float pos1_x = kpsL_x[keypointIdx];
  float pos1_y = kpsL_y[keypointIdx];
  const int border = patchSize / 2;

  // SSD calculation for the current keypoint against all keypoints in
  // keypointsR
  int bestMatchIndex = -1;
  uint64_t bestMatchSSD = 0xffffffffffffffff;
  for (int j = 0; j < numKpsR; ++j) {
    float pos2_x = kpsR_x[j];
    float pos2_y = kpsR_y[j];

    // Compute SSD
    uint64_t ssd = 0;
    for (int dy = -border; dy <= border; ++dy) {
      for (int dx = -border; dx <= border; ++dx) {
        uchar4 p1 = tex2D(texImage1, pos1_x + dx, pos1_y + dy);
        uchar4 p2 = tex2D(texImage2, pos2_x + dx, pos2_y + dy);

        ssd += (p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y) +
               (p1.z - p2.z) * (p1.z - p2.z);
      }
    }

    if (ssd < bestMatchSSD) {
      bestMatchSSD = ssd;
      bestMatchIndex = j;
    }
  }

  // Store the best match for this keypoint
  if (bestMatchSSD < cuMaxSSDThreash) {
    bestMatchIndices[keypointIdx] = bestMatchIndex;
    bestMatchSSDs[keypointIdx] = bestMatchSSD;
  }
}

/**
 * @brief Match keypoints detected by Harris corner detector
 *
 * We typically take a patch around the keypoint and compare the distance
 *
 * @return std::vector<cv::DMatch> the matches
 */
std::vector<cv::DMatch> CudaHarrisKeypointMatcher::matchKeyPoints(
    std::vector<cv::KeyPoint> keypointsL,
    std::vector<cv::KeyPoint> keypointsR) {
  // options
  const int patchSize = options_.patchSize_;
  const uint64_t maxSSDThresh = options_.maxSSDThresh_;

  // copy to __constant__
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cuMaxSSDThreash), &maxSSDThresh,
                                sizeof(uint64_t), 0, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cuPatchSize), &patchSize, sizeof(int), 0,
                                hipMemcpyHostToDevice));

  std::vector<cv::DMatch> matches;

  std::vector<float> pointsL_x(keypointsL.size()), pointsL_y(keypointsL.size());
  std::vector<float> pointsR_x(keypointsR.size()), pointsR_y(keypointsR.size());
  for (size_t i = 0; i < keypointsL.size(); ++i) {
    pointsL_x[i] = keypointsL[i].pt.x;
    pointsL_y[i] = keypointsL[i].pt.y;
  }
  for (size_t i = 0; i < keypointsR.size(); ++i) {
    pointsR_x[i] = keypointsR[i].pt.x;
    pointsR_y[i] = keypointsR[i].pt.y;
  }

  // memory allocation
  float *d_kpsL_X, *d_kpsL_Y, *d_kpsR_X, *d_kpsR_Y;
  CUDA_CHECK(hipMalloc(&d_kpsL_X, keypointsL.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kpsL_Y, keypointsL.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kpsR_X, keypointsR.size() * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_kpsR_Y, keypointsR.size() * sizeof(float)));

  CUDA_CHECK(hipMemcpy(d_kpsL_X, pointsL_x.data(),
                        keypointsL.size() * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_kpsL_Y, pointsL_y.data(),
                        keypointsL.size() * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_kpsR_X, pointsR_x.data(),
                        keypointsR.size() * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_kpsR_Y, pointsR_y.data(),
                        keypointsR.size() * sizeof(float),
                        hipMemcpyHostToDevice));

  // Allocate memory for the best match indices and SSDs on GPU
  int *d_bestMatchIndices;
  uint64_t *d_bestMatchSSDs;
  CUDA_CHECK(hipMalloc(&d_bestMatchIndices, keypointsL.size() * sizeof(int)));
  CUDA_CHECK(
      hipMalloc(&d_bestMatchSSDs, keypointsL.size() * sizeof(uint64_t)));

  // Initialize the best match arrays to -1 (or any sentinel value)
  CUDA_CHECK(
      hipMemset(d_bestMatchIndices, -1, keypointsL.size() * sizeof(int)));
  CUDA_CHECK(
      hipMemset(d_bestMatchSSDs, -1, keypointsL.size() * sizeof(double)));

  int threadsPerBlock = 256;
  int numBlocks = (keypointsL.size() + threadsPerBlock - 1) / threadsPerBlock;

  matchKeypointsKernel<<<numBlocks, threadsPerBlock>>>(
      d_kpsL_X, d_kpsL_Y, d_kpsR_X, d_kpsR_Y, image1_.rows, image1_.cols,
      image2_.rows, image2_.cols, keypointsL.size(), keypointsR.size(),
      d_bestMatchIndices, d_bestMatchSSDs);
  hipError_t kernelError = hipGetLastError();
  if (kernelError != hipSuccess) {
    printf("Kernel Error: %s\n", hipGetErrorString(kernelError));
  }
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy the best match indices and SSDs back to the host
  std::vector<int> bestMatchIndices(keypointsL.size());
  std::vector<double> bestMatchSSDs(keypointsL.size());
  CUDA_CHECK(hipMemcpy(bestMatchIndices.data(), d_bestMatchIndices,
                        keypointsL.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(bestMatchSSDs.data(), d_bestMatchSSDs,
                        keypointsL.size() * sizeof(double),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_kpsL_X));
  CUDA_CHECK(hipFree(d_kpsL_Y));
  CUDA_CHECK(hipFree(d_kpsR_X));
  CUDA_CHECK(hipFree(d_kpsR_Y));
  CUDA_CHECK(hipFree(d_bestMatchIndices));
  CUDA_CHECK(hipFree(d_bestMatchSSDs));

  // get the matches
  for (size_t i = 0; i < keypointsL.size(); ++i) {
    if (bestMatchIndices[i] != -1) {
      matches.push_back(cv::DMatch(i, bestMatchIndices[i], bestMatchSSDs[i]));
    }
  }

  return matches;
}
